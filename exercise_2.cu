
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#ifndef N
#define N 1000000 // 1 << 20
#endif
#define ERROR 0.001
#ifndef THREADS
    #define THREADS 256
#endif
// The same code from lecture1 :)
double mysecond(){  
	struct timeval tp;  
	struct timezone tzp;  
	int i;  i = gettimeofday(&tp,&tzp);  
	return ( 
		(double) tp.tv_sec + (double) tp.tv_usec * 1.e-6 );}
	

void cpu_vector_add( float * x, float * y, int n, float A) {
    for(int i = 0; i < n; i++){
        y[i] = A*x[i] + y[i];
    }
}


__global__ void gpu_vector_add(float * __restrict__ x, float * __restrict__  y, int n, float A){
   
   // Each thread calculates its own index
   int i = blockIdx.x*blockDim.x + threadIdx.x;
   if (i < n) y[i] = A*x[i] + y[i];
	
}

int main(){
    float *a, *b, *cudaOut; 
    double t1, t2, t3, t4, t5, t6;  

    printf("Computing SAXPY with %d elements\n", N);

    // Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    cudaOut = (float*)malloc(sizeof(float)*N);
    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
        cudaOut[i] = 2.0f;
    }
    // Allocate GPU memory
    float *aCuda, *bCuda;
    hipMalloc((void **)&aCuda, sizeof(float)*N);
    hipMalloc((void **)&bCuda, sizeof(float)*N); 



    t1 = mysecond();  
    // Copy data to GPU
    hipMemcpy(aCuda, a, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(bCuda, b, sizeof(float)*N, hipMemcpyHostToDevice);
    // Operating


    // Have a fixed number of threads
    int threads = THREADS;
    // Always an extra group in case of non multiple of threads N
    int groups = N/threads + 1; 
 
    t3 = mysecond();  
    gpu_vector_add<<<groups, threads>>>(aCuda, bCuda, N, 1.0); 
    t4 = mysecond(); 

    // Coying back
    hipMemcpy(cudaOut, bCuda, sizeof(float)*N, hipMemcpyDeviceToHost);
    t2 = mysecond();  
    printf("Computing SAXPY on the GPU in %fs (taking  into account memcpy), %fs (operational)... Done!\n", (t2 - t1), (t4 -t3));

    // Freeing cuda resources

    hipFree(aCuda);
    hipFree(bCuda);    

    t5 = mysecond();  
    cpu_vector_add(a, b, N, 1.0);
    t6 = mysecond();  
    printf("Computing SAXPY on the CPU in %fs… Done!\n", (t6 - t5));
   
    printf("Times: %f,%f,%f\n", (t2 -t1), (t4 - t3), (t6 - t5));

    for(int i = 0; i < N; i++){
        if(abs(b[i] - cudaOut[i]) > ERROR)
        {
            printf("Comparing the output for each implementation, it is incorrect at index %d, %f != %f\n",i,b[i], cudaOut[i]); 
            exit(1);
        }
    }

}
