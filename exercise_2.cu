
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 1000000000 // 1 << 20
#define ERROR 0.001

void cpu_vector_add( float * x, float * y, int n, float A) {
    for(int i = 0; i < n; i++){
        y[i] = A*x[i] + y[i];
    }
}


// The main idea is to make this fails
void cpu_vector2_add(float *x, float *y, int n, float A){
	for(int i = 0; i < n; i++){
		y[i] = A*x[i] + y[i];
	}

}

__global__ void gpu_vector_add(float * __restrict__ x, float * __restrict__  y, int n, float A){
   
   int i = blockIdx.x*blockDim.x + threadIdx.x;
   if (i < n) y[i] = A*x[i] + y[i];
	
}

int main(){
    float *a, *b, *cudaOut; 

    // Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    cudaOut = (float*)malloc(sizeof(float)*N);
    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
        cudaOut[i] = 2.0f;
    }

    // Main function
    cpu_vector_add(a, b, N, 1.0);
    
    // Allocate GPU memory
    float *aCuda, *bCuda;
    hipMalloc((void **)&aCuda, sizeof(float)*N);
    hipMalloc((void **)&bCuda, sizeof(float)*N); 



    // Copy data to GPU
    hipMemcpy(aCuda, a, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(bCuda, cudaOut, sizeof(float)*N, hipMemcpyHostToDevice);
    // Operating

    int threads = 256;
    int groups = N/threads + 1; 
    printf("groups %d, threads %d, N %d, G*T %d \n", groups, threads, N, groups*threads);
 
    gpu_vector_add<<<groups, threads>>>(aCuda, bCuda, N, 1.0); 


hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cudaerr));

    // Coying back
    hipMemcpy(cudaOut, bCuda, sizeof(float)*N, hipMemcpyDeviceToHost);

   // Freeing cuda resources

  hipFree(aCuda);
  hipFree(bCuda);    

    //cpu_vector2_add(a, b2, N, 1.0);
   
    for(int i = 0; i < N; i++){
	if(abs(b[i] - cudaOut[i]) > ERROR)
	{
		printf("Comparing the output for each implementation, it is incorrect at index %d, %f != %f\n",i,b[i], cudaOut[i]); 
                exit(1);
	}
    }

}
