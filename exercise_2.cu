
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define N 1 << 30 + 1 // 1 << 20
#define ERROR 0.001

// The same code from lecture1 :)
double mysecond(){  
	struct timeval tp;  
	struct timezone tzp;  
	int i;  i = gettimeofday(&tp,&tzp);  
	return ( 
		(double) tp.tv_sec + (double) tp.tv_usec * 1.e-6 );}
	

void cpu_vector_add( float * x, float * y, int n, float A) {
    for(int i = 0; i < n; i++){
        y[i] = A*x[i] + y[i];
    }
}


__global__ void gpu_vector_add(float * __restrict__ x, float * __restrict__  y, int n, float A){
   
   // Each thread calculates its own index
   int i = blockIdx.x*blockDim.x + threadIdx.x;
   if (i < n) y[i] = A*x[i] + y[i];
	
}

int main(){
    float *a, *b, *cudaOut; 
    double t1, t2;  


    // Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    cudaOut = (float*)malloc(sizeof(float)*N);
    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
        cudaOut[i] = 2.0f;
    }
    // Allocate GPU memory
    float *aCuda, *bCuda;
    hipMalloc((void **)&aCuda, sizeof(float)*N);
    hipMalloc((void **)&bCuda, sizeof(float)*N); 



    t1 = mysecond();  
    // Copy data to GPU
    hipMemcpy(aCuda, a, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(bCuda, b, sizeof(float)*N, hipMemcpyHostToDevice);
    // Operating


    // Have a fixed number of threads
    int threads = 256;
    // Always an extra group in case of non multiple of threads N
    int groups = N/threads + 1; 
 
    gpu_vector_add<<<groups, threads>>>(aCuda, bCuda, N, 1.0); 
    

    // Coying back
    hipMemcpy(cudaOut, bCuda, sizeof(float)*N, hipMemcpyDeviceToHost);
    t2 = mysecond();  
    printf("Computing SAXPY on the GPU in %f s (taking  into account memcpy)... Done!\n", (t2 - t1));

    // Freeing cuda resources

    hipFree(aCuda);
    hipFree(bCuda);    

    t1 = mysecond();  
    cpu_vector_add(a, b, N, 1.0);
    t2 = mysecond();  
    printf("Computing SAXPY on the CPU in %f s… Done!\n", (t2 - t1));
   
    for(int i = 0; i < N; i++){
        if(abs(b[i] - cudaOut[i]) > ERROR)
        {
            printf("Comparing the output for each implementation, it is incorrect at index %d, %f != %f\n",i,b[i], cudaOut[i]); 
            exit(1);
        }
    }

}
