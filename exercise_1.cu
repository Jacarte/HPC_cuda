#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void cuda_hello(){
    int myId = blockIdx.x*blockDim.x +  threadIdx.x; // thread indexing
    printf("Hello World! My thread is %d\n", myId);
}

int main() {
  cuda_hello<<<1,256>>>();

  hipError_t cudaerr = hipDeviceSynchronize();
  if (cudaerr != hipSuccess)
    printf("kernel launch failed with error \"%s\".\n",hipGetErrorString(cudaerr));

  return 0;
}